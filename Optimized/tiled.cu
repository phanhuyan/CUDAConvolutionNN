#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH  12

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    extern __shared__ float shared_mem[]; // shared memory
    const int sm_width = TILE_WIDTH + K - 1;
    // (void)H_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    #define sm(i2, i1, i0) shared_mem[(i2) * (sm_width * S * sm_width * S) + (i1) * (sm_width * S) + i0]

    // Insert your GPU convolution kernel code here
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;
    int num_tile = ceil(W_out * 1.0 / TILE_WIDTH);
    int h = ty + TILE_WIDTH * (bz / num_tile);
    int w = tx + TILE_WIDTH * (bz % num_tile);
    int m = by;  
    int b = bx;
    int h_base = TILE_WIDTH * (bz / num_tile) * S;
    int w_base = TILE_WIDTH * (bz % num_tile) * S;

    for (int i = ty; i < sm_width; i += TILE_WIDTH) {
        for (int j = tx; j < sm_width; j += TILE_WIDTH) {
            for (int m = 0; m < C; m++) {
                for (int k = 0; k < S; k++) {
                    for (int l = 0; l < S; l++) {
                        int p = i + k*sm_width + h_base;
                        int q = j + l*sm_width + w_base;
                        if (p >= 0 && p < H && q >= 0 && q < W) {
                            sm(m, i + k*sm_width, j + l*sm_width) = in_4d(b, m, p, q);
                        } 
                        else {
                            sm(m, i + k*sm_width, j + l*sm_width) = 0.0f;
                        }
                    }
                }
            }
        }
    }
    __syncthreads();
    if (h < H_out && w < W_out) {
        float acc = 0.0f;
        for (int m_in = 0; m_in < C; m_in++) {
            for (int p = 0; p < K; p++) {
                for (int q = 0; q < K; q++) {
                    acc += in_4d(b, m_in, p + h*S, q + w*S) * mask_4d(m, m_in, p, q);
                }
            }
        }
        out_4d(b, m, h, w) = acc;
    }
    #undef out_4d
    #undef in_4d
    #undef mask_4d
    #undef sm
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    hipMalloc((void**)device_output_ptr, sizeof(float) * B * M * ((H - K)/S + 1) * ((W - K)/S + 1));
    hipMalloc((void**)device_input_ptr, sizeof(float) * B * C * H * W);
    hipMalloc((void**)device_mask_ptr, sizeof(float) * M * C * K * K);

    hipMemcpy(*device_input_ptr, host_input, sizeof(float) * B * C * H * W, hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, sizeof(float) * M * C * K * K, hipMemcpyHostToDevice);

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    int sm_width = TILE_WIDTH + K - 1;
    dim3 dimGrid(B, M, ceil((float)((W - K)/S + 1) / TILE_WIDTH) * ceil((float)((H - K)/S + 1) / TILE_WIDTH));
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    conv_forward_kernel<<<dimGrid, dimBlock, sizeof(float) * sm_width * S * sm_width * S * C>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Copy the output back to host
    hipMemcpy(host_output, device_output, sizeof(float) * B * M * ((H - K)/S + 1) * ((W - K)/S + 1), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}