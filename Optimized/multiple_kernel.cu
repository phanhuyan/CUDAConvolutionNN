#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH1  16
#define TILE_WIDTH2  12
__global__ void conv_forward_kernel1(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    // (void)H_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;
    int num_tile = ceil(W_out * 1.0 / TILE_WIDTH1);
    int h = ty + TILE_WIDTH1 * (bz / num_tile);
    int w = tx + TILE_WIDTH1 * (bz % num_tile);
    int m = by;  
    int b = bx;
    float acc = 0.0f;
    if (h < H_out && w < W_out) {
        for (int c = 0; c < C; c++) {
            for (int p = 0; p < K; p++) {
                for (int q = 0; q < K; q++) {
                    acc += in_4d(b, c, p + h*S, q + w*S) * mask_4d(m, c, p, q);
                }
            }
        }
        out_4d(b, m, h, w) = acc;
    }
    #undef out_4d
    #undef in_4d
    #undef mask_4d
}
__global__ void conv_forward_kernel2(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    // (void)H_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;
    int num_tile = ceil(W_out * 1.0 / TILE_WIDTH2);
    int h = ty + TILE_WIDTH2 * (bz / num_tile);
    int w = tx + TILE_WIDTH2 * (bz % num_tile);
    int m = by;  
    int b = bx;
    float acc = 0.0f;
    if (h < H_out && w < W_out) {
        for (int c = 0; c < C; c++) {
            for (int p = 0; p < K; p++) {
                for (int q = 0; q < K; q++) {
                    acc += in_4d(b, c, p + h*S, q + w*S) * mask_4d(m, c, p, q);
                }
            }
        }
        out_4d(b, m, h, w) = acc;
    }
    #undef out_4d
    #undef in_4d
    #undef mask_4d
}
	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    hipMalloc((void**)device_output_ptr, sizeof(float) * B * M * ((H - K)/S + 1) * ((W - K)/S + 1));
    hipMalloc((void**)device_input_ptr, sizeof(float) * B * C * H * W);
    hipMalloc((void**)device_mask_ptr, sizeof(float) * M * C * K * K);

    hipMemcpy(*device_input_ptr, host_input, sizeof(float) * B * C * H * W, hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, sizeof(float) * M * C * K * K, hipMemcpyHostToDevice);

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    if (C == 1){
        dim3 dimGrid1(B, M, ceil((float)((W - K)/S + 1) / TILE_WIDTH1) * ceil((float)((H - K)/S + 1) / TILE_WIDTH1));
        dim3 dimBlock1(TILE_WIDTH1, TILE_WIDTH1, 1);
        conv_forward_kernel1<<<dimGrid1, dimBlock1>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
    } else {
        dim3 dimGrid2(B, M, ceil((float)((W - K)/S + 1) / TILE_WIDTH2) * ceil((float)((H - K)/S + 1) / TILE_WIDTH2));
        dim3 dimBlock2(TILE_WIDTH2, TILE_WIDTH2, 1);
        conv_forward_kernel2<<<dimGrid2, dimBlock2>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
    }
    
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Copy the output back to host
    hipMemcpy(host_output, device_output, sizeof(float) * B * M * ((H - K)/S + 1) * ((W - K)/S + 1), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}